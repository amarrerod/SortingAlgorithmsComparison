/*
 * @Author: Alejandro Marrero 
 * @Contact: alu0100825008@ull.edu.es 
 * @Date: 2018-05-08 18:56:58 
 * @Last Modified time: 2018-05-08 18:56:58 
 */
#include <iostream>
#include <vector>
#include <ctime>
#include <cstdlib>
#include <boost/chrono.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

const int LOWER = 0.0;
const int UPPER = 1.0;

void fillVector(thrust::host_vector<double> &V, const double lower, const double upper, const unsigned int seed) {
    srand(time(NULL));
    size_t elem = V.size();
    for( size_t i = 0; i < elem; ++i){
        V[i] = (double) rand() / (double) RAND_MAX;
    }
}

int main() {
    thrust::host_vector<double> hostVector;
    thrust::device_vector<double> deviceVector;
    unsigned int seed = (unsigned int) time(NULL);
    size_t limit = 100000000;
    for(int i = 500; i <= limit; i *= 2 ) {
        hostVector.resize(i);
        fillVector(V, LOWER, UPPER, seed);
        boost::chrono::steady_clock::time_point start_cpu = boost::chrono::steady_clock::now();
        deviceVector = hostVector;
        boost::chrono::steady_clock::time_point end_cpu = boost::chrono::steady_clock::now();
        double durationCPU = boost::chrono::duration <double, boost::milli> (end_cpu - start_cpu).count();
	    hipEvent_t start, stop;
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventRecord(start,0);
        thrust::stable_sort(d_V.begin(), d_V.end());
	    hipEventRecord(stop,0);
	    hipEventSynchronize(stop);
	    float elapsedTime;
	    hipEventElapsedTime(&elapsedTime, start, stop);
	    hipEventDestroy(start);
	    hipEventDestroy(stop);
        start_cpu = boost::chrono::steady_clock::now();
        hostVector = deviceVector;
        end_cpu = boost::chrono::steady_clock::now();
        double durationCPU_2 = boost::chrono::duration <double, boost::milli> (end_cpu - start_cpu).count();
	    std::cout << i << std::endl << ((elapsedTime + durationCPU + durationCPU_2) * 0.001) << std::endl;
    }
    return 0;
}

