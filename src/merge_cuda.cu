#include <iostream>
#include <vector>
#include <ctime>
#include <cstdlib>
#include <boost/chrono.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

void rnd_fill(thrust::host_vector<double> &V, const double lower, const double upper, const unsigned int seed) {
    srand(time(NULL));
    size_t elem = V.size();
    for( size_t i = 0; i < elem; ++i){
        V[i] = (double) rand() / (double) RAND_MAX;
    }
}

int main() {
    thrust::host_vector<double> V;
    thrust::device_vector<double> d_V;
    //use the system time to create a random seed
    unsigned int seed = (unsigned int) time(NULL);
    size_t mem = 100000000;
    for(int i = 500; i <= mem; i *= 2 ) {
        V.resize(i);
        rnd_fill(V, -100.0, 100.0, seed);
        boost::chrono::steady_clock::time_point start_cpu = boost::chrono::steady_clock::now();
        d_V = V; // Transfer data to the GPU
        boost::chrono::steady_clock::time_point end_cpu = boost::chrono::steady_clock::now();
        double dt1 = boost::chrono::duration <double, boost::milli> (end_cpu - start_cpu).count();
	    hipEvent_t start, stop;
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    //Start recording
	    hipEventRecord(start,0);
        thrust::stable_sort(d_V.begin(), d_V.end());
	    //Stop recording
	    hipEventRecord(stop,0);
	    hipEventSynchronize(stop);
	    float elapsedTime;
	    hipEventElapsedTime(&elapsedTime, start, stop);
	    hipEventDestroy(start);
	    hipEventDestroy(stop);
        start_cpu = boost::chrono::steady_clock::now();
        V = d_V; // Transfer data to the CPU
        end_cpu = boost::chrono::steady_clock::now();
        double dt2 = boost::chrono::duration <double, boost::milli> (end_cpu - start_cpu).count();
	    std::cout << i << std::endl << ((elapsedTime + dt1 + dt2) * 0.001) << std::endl;
    }
    return 0;
}

